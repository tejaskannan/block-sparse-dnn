
#include <hip/hip_runtime.h>
#ifdef GOOGLE_CUDA
#define EIGEN_USE_GPU
#include "bsmm.h"
#include "tensorflow/core/util/gpu_kernel_helper.h"
#include "cuda_runtime.h"
#include "cuda.h"


using namespace tensorflow;

using GPUDevice = Eigen::GpuDevice;

template<class T>
struct shared_memory
{
  __device__ inline operator T *()
  {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }

  __device__ inline operator const T *() const
  {
    extern __shared__ int __smem[];
    return (T *)__smem;
  }
};

template <typename index_type>
__device__ index_type round_up_to_power_of_two (index_type v)
{
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;

  return v;
}

// CUDA kernel based on https://medium.com/gpgpu/block-sparse-matrix-vector-multiplication-with-cuda-4e616b30267
template <typename data_type, typename index_type>
__global__ void bcsr_spmv_kernel_column_by_column(
  const index_type bs,
  const index_type * __restrict__ col_ids,
  const index_type * __restrict__ row_ptr,
  const data_type * __restrict__ data,
  const data_type * __restrict__ x,
  data_type * __restrict__ y)
{
  const index_type idx = blockIdx.x * blockDim.x + threadIdx.x;
  const index_type lane = idx % 32;
  const index_type block_row = idx / 32; ///< Warp per block row
  const index_type first_block = row_ptr[block_row];
  const index_type last_block = row_ptr[block_row + 1];

  index_type col = first_block * bs + lane / bs;
  index_type r = lane % bs;

  data_type *partial_sums = shared_memory<data_type> (); ///< Size is equal to blockDim.x * sizeof(data_type)

  data_type local_out = 0.0;

  for (; col < last_block * bs; col += 32 / bs)
    {
      const index_type block = col / bs;
      const index_type c = col % bs;

      const data_type value = data[block * bs * bs + c * bs + r];
      const data_type x_value = x[col_ids[block] * bs + c];
      local_out += x_value * value;
    }

  partial_sums[threadIdx.x] = local_out;

  for (index_type stride = round_up_to_power_of_two((32 / bs) / 2); stride > 0; stride /= 2)
    {
      __syncthreads ();
      if ((lane < stride * bs) && ((threadIdx.x + stride * bs) < 32))
        partial_sums[threadIdx.x] += partial_sums[threadIdx.x + stride * bs];
    }

  if (lane < bs)
    y[block_row * bs + lane] = partial_sums[threadIdx.x];
}

template<typename DataType, typename IndexType>
void BCSRMatMulFunctor<GPUDevice, DataType, IndexType>::operator()(const GPUDevice& d, int block_size, IndexType* col_ids, IndexType* row_ptr, DataType* blocks, DataType* dense, DataType* out) {
        int block_count = 1024;
        int thread_per_block = 20;

        bcsr_spmv_kernel_column_by_column<DataType, IndexType><<<block_count, thread_per_block, 0, d.stream()>>>(block_size, col_ids, row_ptr, blocks, dense, out);
    }


// Explicitly instantiate functors for the types of OpKernels registered.
template struct BCSRMatMulFunctor<GPUDevice, float, uint64>;

#endif  // GOOGLE_CUDA
